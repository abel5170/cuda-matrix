#include <hip/hip_runtime.h>
#include <iostream>

static inline void checkCuda(hipError_t err, const char* msg = "") {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " : " << hipGetErrorString(err) << std::endl;
        std::abort();
    }
}

__global__ void matMulKernel(const double* A, const double* B, double* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        double sum = 0.0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// wrapper: times kernel using cuda events and returns elapsed ms in gpu_ms
extern "C" void gpu_multiply_raw(const double* Arow, const double* Brow, double* Crow, int N, float &gpu_ms) {
    size_t bytes = sizeof(double) * size_t(N) * size_t(N);
    double *dA=nullptr, *dB=nullptr, *dC=nullptr;
    checkCuda(hipMalloc((void**)&dA, bytes), "hipMalloc dA");
    checkCuda(hipMalloc((void**)&dB, bytes), "hipMalloc dB");
    checkCuda(hipMalloc((void**)&dC, bytes), "hipMalloc dC");

    checkCuda(hipMemcpy(dA, Arow, bytes, hipMemcpyHostToDevice), "H2D A");
    checkCuda(hipMemcpy(dB, Brow, bytes, hipMemcpyHostToDevice), "H2D B");

    dim3 block(16,16);
    dim3 grid( (N + block.x - 1)/block.x, (N + block.y - 1)/block.y );

    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start), "Create start");
    checkCuda(hipEventCreate(&stop), "Create stop");
    checkCuda(hipEventRecord(start), "Record start");

    matMulKernel<<<grid, block>>>(dA, dB, dC, N);
    // check kernel
    checkCuda(hipGetLastError(), "Kernel launch");

    checkCuda(hipEventRecord(stop), "Record stop");
    checkCuda(hipEventSynchronize(stop), "Synchronize stop");
    checkCuda(hipEventElapsedTime(&gpu_ms, start, stop), "ElapsedTime");

    checkCuda(hipMemcpy(Crow, dC, bytes, hipMemcpyDeviceToHost), "D2H C");

    checkCuda(hipEventDestroy(start), "Destroy start");
    checkCuda(hipEventDestroy(stop), "Destroy stop");
    hipFree(dA); hipFree(dB); hipFree(dC);
}
